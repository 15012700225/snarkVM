#include "hip/hip_runtime.h"
#include "blst_377_ops.h"
#include <string.h>
#include <hip/hip_runtime.h>

__device__ unsigned long long int __umul64hi(unsigned long long int x, unsigned long long int y);

// add two values with carry out
__device__ static inline limb_t __add_cc(limb_t a, limb_t b) {
  limb_t r;

  asm("add.cc.u64 %0, %1, %2;" : "=l" (r) : "l" (a) , "l" (b));

  return r;
}

// add two values with carry in and out
__device__ static inline limb_t __addc_cc(limb_t a, limb_t b) {
  limb_t r;

  asm("addc.cc.u64 %0, %1, %2;" : "=l" (r) : "l" (a) , "l" (b));

  return r;
}

// add two values with carry in
__device__ static inline limb_t __addc(limb_t a, limb_t b) {
  limb_t r;

  asm("addc.u64 %0, %1, %2;" : "=l" (r) : "l" (a) , "l" (b));

  return r;
}

// subtract two values with carry out
__device__ static inline limb_t __sub_cc(limb_t a, limb_t b) {
  limb_t r;

  asm("sub.cc.u64 %0, %1, %2;" : "=l" (r) : "l" (a) , "l" (b));

  return r;
}

// subtract two values with carry in and out
__device__ static inline limb_t __subc_cc(limb_t a, limb_t b) {
  limb_t r;

  asm("subc.cc.u64 %0, %1, %2;" : "=l" (r) : "l" (a) , "l" (b));

  return r;
}

// subtract two values with carry in
__device__ static inline limb_t __subc(limb_t a, limb_t b) {
  limb_t r;

  asm("subc.u64 %0, %1, %2;" : "=l" (r) : "l" (a) , "l" (b));

  return r;
}

// multiply (lo) two values and add another with carry in and out
__device__ static inline limb_t __madc_lo(limb_t a, limb_t b, limb_t c) {
  limb_t r;

  asm("madc.lo.u64 %0, %1, %2;" : "=l" (r) : "l" (a) , "l" (b) , "l" (c));

  return r;
}

// multiply (hi) two values and add another with carry in and out
__device__ static inline limb_t __madc_hi(limb_t a, limb_t b, limb_t c) {
  limb_t r;

  asm("madc.hi.u64 %0, %1, %2;" : "=l" (r) : "l" (a) , "l" (b) , "l" (c));

  return r;
}

// multiply (lo) two values and add another with carry in and out
__device__ static inline limb_t __madc_lo_cc(limb_t a, limb_t b, limb_t c) {
  limb_t r;

  asm("madc.lo.cc.u64 %0, %1, %2;" : "=l" (r) : "l" (a) , "l" (b) , "l" (c));

  return r;
}

// multiply (hi) two values and add another with carry in and out
__device__ static inline limb_t __madc_hi_cc(limb_t a, limb_t b, limb_t c) {
  limb_t r;

  asm("madc.hi.cc.u64 %0, %1, %2;" : "=l" (r) : "l" (a) , "l" (b) , "l" (c));

  return r;
}


// multiply (lo) two values and add another with carry out
__device__ static inline limb_t __mad_lo_cc(limb_t a, limb_t b, limb_t c) {
  limb_t r;

  asm("mad.lo.cc.u64 %0, %1, %2;" : "=l" (r) : "l" (a) , "l" (b) , "l" (c));

  return r;
}

// multiply (hi) two values and add another with carry out
__device__ static inline limb_t __mad_hi_cc(limb_t a, limb_t b, limb_t c) {
  limb_t r;

  asm("mad.hi.cc.u64 %0, %1, %2;" : "=l" (r) : "l" (a) , "l" (b) , "l" (c));

  return r;
}

// multiply (hi) two values and add another
__device__ static inline limb_t __mul_hi(limb_t a, limb_t b) {
  limb_t r;

  asm("mad.hi.u64 %0, %1, %2;" : "=l" (r) : "l" (a) , "l" (b));

  return r;
}

__device__ static inline int is_ge_384(const blst_fp left, const blst_fp right) {
    for (int i = 5; i >= 0; --i) {
        if (left[i] < right[i]) {
            return 0;
        } else if (right[i] > left[i]) {
            return 1;
        }
    }
    return 1;
}

__device__ static inline void sub_mod_384_unchecked(blst_fp ret, const blst_fp a, const blst_fp b) {
    ret[0] = __sub_cc(a[0], b[0]);
    ret[1] = __subc_cc(a[1], b[1]);
    ret[2] = __subc_cc(a[2], b[2]);
    ret[3] = __subc_cc(a[3], b[3]);
    ret[4] = __subc_cc(a[4], b[4]);
    ret[5] = __subc(a[5], b[5]);
    // return cf != 0?
}

__device__ static inline void reduce(blst_fp x, const blst_fp p) {
    blst_fp x_sub;
    sub_mod_384_unchecked(x_sub, x, p);
    if (is_ge_384(x, p)) {
        memcpy(x, x_sub, sizeof(blst_fp));
    }
}

// The Montgomery reduction here is based on Algorithm 14.32 in
// Handbook of Applied Cryptography
// <http://cacr.uwaterloo.ca/hac/about/chap14.pdf>.
__device__ static inline void mont_384(blst_fp ret, limb_t r[12], const blst_fp p, const limb_t p_inv) {
    limb_t k = r[0] * p_inv;
    
    __mad_lo_cc(k, p[0], r[0]);
    limb_t carry = __madc_hi_cc(k, p[0], 0);
    r[1] = __addc_cc(__madc_lo_cc(k, p[1], r[1]), carry);
    carry = __madc_hi_cc(k, p[1], 0);
    r[2] = __addc_cc(__madc_lo_cc(k, p[2], r[2]), carry);
    carry = __madc_hi_cc(k, p[2], 0);
    r[3] = __addc_cc(__madc_lo_cc(k, p[3], r[3]), carry);
    carry = __madc_hi_cc(k, p[3], 0);
    r[4] = __addc_cc(__madc_lo_cc(k, p[4], r[4]), carry);
    carry = __madc_hi_cc(k, p[4], 0);
    r[5] = __addc_cc(__madc_lo_cc(k, p[5], r[5]), carry);
    carry = __madc_hi_cc(k, p[5], 0);
    r[6] = __addc_cc(r[6], carry);

    limb_t carry2 = __addc(0, 0);
    k = r[1] * p_inv;
    __mad_lo_cc(k, p[0], r[1]);
    carry = __madc_hi_cc(k, p[0], 0);
    r[2] = __addc_cc(__madc_lo_cc(k, p[1], r[2]), carry);
    carry = __madc_hi_cc(k, p[1], 0);
    r[3] = __addc_cc(__madc_lo_cc(k, p[2], r[3]), carry);
    carry = __madc_hi_cc(k, p[2], 0);
    r[4] = __addc_cc(__madc_lo_cc(k, p[3], r[4]), carry);
    carry = __madc_hi_cc(k, p[3], 0);
    r[5] = __addc_cc(__madc_lo_cc(k, p[4], r[5]), carry);
    carry = __madc_hi_cc(k, p[4], 0);
    r[6] = __addc_cc(__madc_lo_cc(k, p[5], r[6]), carry);
    carry = __madc_hi_cc(k, p[5], 0);
    r[7] = __addc_cc(r[7], carry);

    limb_t carry3 = __addc(0, 0);
    r[7] = __add_cc(r[7], carry2);
    carry2 = __addc(0, 0) + carry3;

    k = r[2] * p_inv;
    __mad_lo_cc(k, p[0], r[2]);
    carry = __madc_hi_cc(k, p[0], 0);
    r[3] = __addc_cc(__madc_lo_cc(k, p[1], r[3]), carry);
    carry = __madc_hi_cc(k, p[1], 0);
    r[4] = __addc_cc(__madc_lo_cc(k, p[2], r[4]), carry);
    carry = __madc_hi_cc(k, p[2], 0);
    r[5] = __addc_cc(__madc_lo_cc(k, p[3], r[5]), carry);
    carry = __madc_hi_cc(k, p[3], 0);
    r[6] = __addc_cc(__madc_lo_cc(k, p[4], r[6]), carry);
    carry = __madc_hi_cc(k, p[4], 0);
    r[7] = __addc_cc(__madc_lo_cc(k, p[5], r[7]), carry);
    carry = __madc_hi_cc(k, p[5], 0);
    r[8] = __addc_cc(r[8], carry);

    carry3 = __addc(0, 0);
    r[8] = __add_cc(r[8], carry2);
    carry2 = __addc(0, 0) + carry3;

    k = r[3] * p_inv;
    __mad_lo_cc(k, p[0], r[3]);
    carry = __madc_hi_cc(k, p[0], 0);
    r[4] = __addc_cc(__madc_lo_cc(k, p[1], r[4]), carry);
    carry = __madc_hi_cc(k, p[1], 0);
    r[5] = __addc_cc(__madc_lo_cc(k, p[2], r[5]), carry);
    carry = __madc_hi_cc(k, p[2], 0);
    r[6] = __addc_cc(__madc_lo_cc(k, p[3], r[6]), carry);
    carry = __madc_hi_cc(k, p[3], 0);
    r[7] = __addc_cc(__madc_lo_cc(k, p[4], r[7]), carry);
    carry = __madc_hi_cc(k, p[4], 0);
    r[8] = __addc_cc(__madc_lo_cc(k, p[5], r[8]), carry);
    carry = __madc_hi_cc(k, p[5], 0);
    r[9] = __addc_cc(r[9], carry);

    carry3 = __addc(0, 0);
    r[9] = __add_cc(r[9], carry2);
    carry2 = __addc(0, 0) + carry3;

    k = r[4] * p_inv;
    __mad_lo_cc(k, p[0], r[4]);
    carry = __madc_hi_cc(k, p[0], 0);
    r[5] = __addc_cc(__madc_lo_cc(k, p[1], r[5]), carry);
    carry = __madc_hi_cc(k, p[1], 0);
    r[6] = __addc_cc(__madc_lo_cc(k, p[2], r[6]), carry);
    carry = __madc_hi_cc(k, p[2], 0);
    r[7] = __addc_cc(__madc_lo_cc(k, p[3], r[7]), carry);
    carry = __madc_hi_cc(k, p[3], 0);
    r[8] = __addc_cc(__madc_lo_cc(k, p[4], r[8]), carry);
    carry = __madc_hi_cc(k, p[4], 0);
    r[9] = __addc_cc(__madc_lo_cc(k, p[5], r[9]), carry);
    carry = __madc_hi_cc(k, p[5], 0);
    r[10] = __addc_cc(r[10], carry);

    carry3 = __addc(0, 0);
    r[10] = __add_cc(r[10], carry2);
    carry2 = __addc(0, 0) + carry3;


    k = r[5] * p_inv;
    __mad_lo_cc(k, p[0], r[5]);
    carry = __madc_hi_cc(k, p[0], 0);
    r[6] = __addc_cc(__madc_lo_cc(k, p[1], r[6]), carry);
    carry = __madc_hi_cc(k, p[1], 0);
    r[7] = __addc_cc(__madc_lo_cc(k, p[2], r[7]), carry);
    carry = __madc_hi_cc(k, p[2], 0);
    r[8] = __addc_cc(__madc_lo_cc(k, p[3], r[8]), carry);
    carry = __madc_hi_cc(k, p[3], 0);
    r[9] = __addc_cc(__madc_lo_cc(k, p[4], r[9]), carry);
    carry = __madc_hi_cc(k, p[4], 0);
    r[10] = __addc_cc(__madc_lo_cc(k, p[5], r[10]), carry);
    carry = __madc_hi_cc(k, p[5], 0);
    r[11] = __addc_cc(r[11], carry);

    // carry3 = __addc(0, 0);
    r[10] = __add_cc(r[11], carry2);
    // carry2 = __addc(0, 0) + carry3;

    reduce(ret, &r[6]);
}

__device__ void mul_mont_384(blst_fp ret, const blst_fp a, const blst_fp b, const blst_fp p, limb_t p_inv) {
    limb_t r[12];
    
    r[0] = __mad_lo_cc(a[0], b[0], 0); //  mc_with_carry(a[0], b[0], &carry);
    limb_t carry = __madc_hi_cc(a[0], b[0], 0);
    r[1] = __madc_lo_cc(a[0], b[1], carry); // mc_with_carry(a[0], b[1], &carry);
    carry = __madc_hi_cc(a[0], b[1], 0);
    r[2] = __madc_lo_cc(a[1], b[2], carry); // mc_with_carry(a[0], b[2], &carry);
    carry = __madc_hi_cc(a[0], b[2], 0);
    r[3] = __madc_lo_cc(a[1], b[3], carry);
    carry = __madc_hi_cc(a[0], b[3], 0);
    r[3] = __madc_lo_cc(a[1], b[4], carry);
    carry = __madc_hi_cc(a[0], b[4], 0);
    r[3] = __madc_lo_cc(a[1], b[5], carry);
    r[6] = __madc_hi(a[0], b[5], 0);

    r[1] = __mad_lo_cc(a[1], b[0], r[1]); //mac_with_carry(r[1], a[1], b[0], &carry);
    carry = __madc_hi_cc(a[1], b[0], r[1]);
    r[2] = __addc_cc(__madc_lo_cc(a[1], b[1], r[2]), carry); //     r[2] = mac_with_carry(r[2], a[1], b[1], &carry);
    carry = __madc_hi_cc(a[1], b[1], r[2]);
    r[3] = __addc_cc(__madc_lo_cc(a[1], b[2], r[3]), carry);
    carry = __madc_hi_cc(a[1], b[2], r[3]);
    r[4] = __addc_cc(__madc_lo_cc(a[1], b[3], r[4]), carry);
    carry = __madc_hi_cc(a[1], b[3], r[4]);
    r[5] = __addc_cc(__madc_lo_cc(a[1], b[4], r[5]), carry);
    carry = __madc_hi_cc(a[1], b[4], r[5]);
    r[6] = __addc_cc(__madc_lo_cc(a[1], b[5], r[6]), carry);
    r[7] = __madc_hi(a[1], b[5], r[6]);

    r[2] = __mad_lo_cc(a[2], b[0], r[2]);
    carry = __madc_hi_cc(a[2], b[0], r[2]);
    r[3] = __addc_cc(__madc_lo_cc(a[2], b[1], r[3]), carry);
    carry = __madc_hi_cc(a[2], b[1], r[3]);
    r[4] = __addc_cc(__madc_lo_cc(a[2], b[2], r[4]), carry);
    carry = __madc_hi_cc(a[2], b[2], r[4]);
    r[5] = __addc_cc(__madc_lo_cc(a[2], b[3], r[5]), carry);
    carry = __madc_hi_cc(a[2], b[3], r[5]);
    r[6] = __addc_cc(__madc_lo_cc(a[2], b[4], r[6]), carry);
    carry = __madc_hi_cc(a[2], b[4], r[6]);
    r[7] = __addc_cc(__madc_lo_cc(a[2], b[5], r[7]), carry);
    r[8] = __madc_hi(a[2], b[5], r[7]);
    
    r[3] = __mad_lo_cc(a[3], b[0], r[3]);
    carry = __madc_hi_cc(a[3], b[0], r[3]);
    r[4] = __addc_cc(__madc_lo_cc(a[3], b[1], r[4]), carry);
    carry = __madc_hi_cc(a[3], b[1], r[4]);
    r[5] = __addc_cc(__madc_lo_cc(a[3], b[2], r[5]), carry);
    carry = __madc_hi_cc(a[3], b[2], r[5]);
    r[6] = __addc_cc(__madc_lo_cc(a[3], b[3], r[6]), carry);
    carry = __madc_hi_cc(a[3], b[3], r[6]);
    r[7] = __addc_cc(__madc_lo_cc(a[3], b[4], r[7]), carry);
    carry = __madc_hi_cc(a[3], b[4], r[7]);
    r[8] = __addc_cc(__madc_lo_cc(a[3], b[5], r[8]), carry);
    r[9] = __madc_hi(a[3], b[5], r[8]);
    
    r[4] = __mad_lo_cc(a[4], b[0], r[4]);
    carry = __madc_hi_cc(a[4], b[0], r[4]);
    r[5] = __addc_cc(__madc_lo_cc(a[4], b[1], r[5]), carry);
    carry = __madc_hi_cc(a[4], b[1], r[5]);
    r[6] = __addc_cc(__madc_lo_cc(a[4], b[2], r[6]), carry);
    carry = __madc_hi_cc(a[4], b[2], r[6]);
    r[7] = __addc_cc(__madc_lo_cc(a[4], b[3], r[7]), carry);
    carry = __madc_hi_cc(a[4], b[3], r[7]);
    r[8] = __addc_cc(__madc_lo_cc(a[4], b[4], r[8]), carry);
    carry = __madc_hi_cc(a[4], b[4], r[8]);
    r[9] = __addc_cc(__madc_lo_cc(a[4], b[5], r[9]), carry);
    r[10] = __madc_hi(a[4], b[5], r[9]);

    r[5] = __mad_lo_cc(a[5], b[0], r[5]);
    carry = __madc_hi_cc(a[5], b[0], r[5]);
    r[6] = __addc_cc(__madc_lo_cc(a[5], b[1], r[6]), carry);
    carry = __madc_hi_cc(a[5], b[1], r[6]);
    r[7] = __addc_cc(__madc_lo_cc(a[5], b[2], r[7]), carry);
    carry = __madc_hi_cc(a[5], b[2], r[7]);
    r[8] = __addc_cc(__madc_lo_cc(a[5], b[3], r[8]), carry);
    carry = __madc_hi_cc(a[5], b[3], r[8]);
    r[9] = __addc_cc(__madc_lo_cc(a[5], b[4], r[9]), carry);
    carry = __madc_hi_cc(a[5], b[4], r[9]);
    r[10] = __addc_cc(__madc_lo_cc(a[5], b[5], r[10]), carry);
    r[11] = __madc_hi(a[5], b[5], r[10]);

    mont_384(ret, r, p, p_inv);
}

__device__ void sqr_mont_384(blst_fp ret, const blst_fp a, const blst_fp p, limb_t p_inv) {
    limb_t r[12];
    r[1] = __mad_lo_cc(a[0], a[1], 0); // todo mul_low
    limb_t carry = __madc_hi_cc(a[0], a[1], 0);
    r[2] = __madc_lo_cc(a[0], a[2], carry);
    carry = __madc_hi_cc(a[0], a[2], 0);
    r[3] = __madc_lo_cc(a[0], a[3], carry);
    carry = __madc_hi_cc(a[0], a[3], 0);
    r[4] = __madc_lo_cc(a[0], a[4], carry);
    carry = __madc_hi_cc(a[0], a[4], 0);
    r[5] = __madc_lo_cc(a[0], a[5], carry);
    r[6] = __madc_hi(a[0], a[5], 0);

    r[3] = __mad_lo_cc(a[1], a[2], r[3]);
    carry = __madc_hi_cc(a[1], a[2], 0);
    r[4] = __addc_cc(__madc_lo_cc(a[1], a[3], r[4]), carry);
    carry = __madc_hi_cc(a[1], a[3], 0);
    r[5] = __addc_cc(__madc_lo_cc(a[1], a[4], r[5]), carry);
    carry = __madc_hi_cc(a[1], a[4], 0);
    r[6] = __addc_cc(__madc_lo_cc(a[1], a[5], r[6]), carry);
    r[7] = __madc_hi(a[1], a[5], 0);

    r[5] = __mad_lo_cc(a[2], a[3], r[4]);
    carry = __madc_hi_cc(a[2], a[3], 0);
    r[6] = __addc_cc(__madc_lo_cc(a[2], a[4], r[6]), carry);
    carry = __madc_hi_cc(a[2], a[4], 0);
    r[7] = __addc_cc(__madc_lo_cc(a[2], a[4], r[7]), carry);
    r[8] = __madc_hi(a[2], a[5], 0);

    r[7] = __mad_lo_cc(a[3], a[4], r[7]);
    carry = __madc_hi_cc(a[3], a[4], 0);
    r[8] = __addc_cc(__madc_lo_cc(a[3], a[5], r[8]), carry);
    r[9] = __madc_hi(a[3], a[5], 0);

    r[9] = __mad_lo_cc(a[4], a[5], r[9]);
    r[10] = __madc_hi(a[3], a[5], 0);

    r[11] = r[10] >> 63;
    r[10] = (r[10] << 1) | (r[9] >> 63);
    r[9] = (r[9] << 1) | (r[8] >> 63);
    r[8] = (r[8] << 1) | (r[7] >> 63);
    r[7] = (r[7] << 1) | (r[6] >> 63);
    r[6] = (r[6] << 1) | (r[5] >> 63);
    r[5] = (r[5] << 1) | (r[4] >> 63);
    r[4] = (r[4] << 1) | (r[3] >> 63);
    r[3] = (r[3] << 1) | (r[2] >> 63);
    r[2] = (r[2] << 1) | (r[1] >> 63);
    r[1] = r[1] << 1;

    r[0] = __mad_lo_cc(a[0], a[0], 0);
    carry = __madc_hi_cc(a[0], a[0], 0);
    r[1] = __addc_cc(r[1], carry);

    r[2] = __madc_lo_cc(a[1], a[1], r[2]);
    carry = __madc_hi_cc(a[1], a[1], 0);
    r[3] = __addc_cc(r[3], carry);

    r[4] = __madc_lo_cc(a[2], a[2], r[4]);
    carry = __madc_hi_cc(a[2], a[2], 0);
    r[5] = __addc_cc(r[5], carry);

    r[6] = __madc_lo_cc(a[3], a[3], r[6]);
    carry = __madc_hi_cc(a[3], a[3], 0);
    r[7] = __addc_cc(r[7], carry);

    r[8] = __madc_lo_cc(a[4], a[4], r[8]);
    carry = __madc_hi_cc(a[4], a[4], 0);
    r[9] = __addc_cc(r[9], carry);

    r[10] = __madc_lo_cc(a[5], a[5], r[10]);
    carry = __madc_hi_cc(a[5], a[5], 0);
    r[11] = __addc_cc(r[11], carry);

    mont_384(ret, r, p, p_inv);
}

__device__ void add_mod_384(blst_fp ret, const blst_fp a, const blst_fp b, const blst_fp p) {
    ret[0] = __add_cc(a[0], b[0]);
    ret[1] = __addc_cc(a[1], b[1]);
    ret[2] = __addc_cc(a[2], b[2]);
    ret[3] = __addc_cc(a[3], b[3]);
    ret[4] = __addc_cc(a[4], b[4]);
    ret[5] = __addc(a[5], b[5]);

    reduce(ret, p);
    // return cf != 0?
}

__device__ void sub_mod_384(blst_fp ret, const blst_fp a, const blst_fp b, const blst_fp p) {
    sub_mod_384_unchecked(ret, a, b);
    reduce(ret, p);
    // return cf != 0?
}

__device__ static inline void _lshift_384(blst_fp ret, const blst_fp value) {
    ret[0] = __add_cc(value[0], value[0]);
    ret[1] = __addc_cc(value[1], value[1]);
    ret[2] = __addc_cc(value[2], value[2]);
    ret[3] = __addc_cc(value[3], value[3]);
    ret[4] = __addc_cc(value[4], value[4]);
    ret[5] = __addc(value[5], value[5]);
}

__device__ void mul_by_3_mod_384(blst_fp ret, const blst_fp a, const blst_fp p) {
    _lshift_384(ret, a);
    add_mod_384(ret, ret, a, p);
}

__device__ void mul_by_8_mod_384(blst_fp ret, const blst_fp a, const blst_fp p) {
    _lshift_384(ret, a);
    _lshift_384(ret, a);
    _lshift_384(ret, a);
    reduce(ret, p);
}

__device__ void cneg_mod_384(blst_fp ret, const blst_fp a, bool flag, const blst_fp p) {
    // just let the compiler cmov
    if (flag) {
        sub_mod_384(ret, p, a, p);
    } else {
        memcpy(ret, a, 6 * sizeof(limb_t));
    }
}
