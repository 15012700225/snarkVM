#include "hip/hip_runtime.h"
#include "types.h"
#include "asm_cuda.h"
#include "blst_377_ops.h"

extern "C" __global__ void sqr_test(blst_fp ret, const blst_fp a) {
    blst_fp_sqr(ret, a);
}

extern "C" __global__ void add_test(blst_fp ret, const blst_fp* a) {
    blst_fp_add(ret, a[0], a[1]);
}

extern "C" __global__ void add_projective_test(blst_p1* ret, const blst_p1* a) {
    blstv2_add_projective_to_projective(ret, &a[0], &a[1]);
}

extern "C" __global__ void sub_test(blst_fp ret, const blst_fp* a) {
    blst_fp_sub(ret, a[0], a[1]);
}

extern "C" __global__ void mul_test(blst_fp ret, const blst_fp* a) {
    blst_fp_mul(ret, a[0], a[1]);
}

extern "C" __global__ void div2_test(blst_fp ret, const blst_fp a) {
    div_by_2_mod_384(ret, a);
}

extern "C" __global__ void inverse_test(blst_fp ret, const blst_fp a) {
    blst_inverse(ret, a);
}

