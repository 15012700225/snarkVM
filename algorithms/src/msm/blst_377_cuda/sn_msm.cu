#include "hip/hip_runtime.h"
// Copyright Supranational LLC
// Licensed under the Apache License, Version 2.0, see LICENSE for details.
// SPDX-License-Identifier: Apache-2.0

#include <algorithm>
#include <iostream>
#include <cstring>
#include <immintrin.h>
#include "sn_msm.h"
#include <cmath>

/****************************************************************************
 * #6. Pippenger - Gurvy, min branches, Jac Extended, Rust compatible
 ****************************************************************************/
void msm_pippenger_6(blst_p1* result,
                     const rust_p1_affine* bases_in,
                     const blst_scalar* scalars_in,
                     size_t num_pairs,
                     size_t scalar_bits,
                     size_t c) {
  // Normally c (window size) is calculated or empirically set
  // Allow as input to improve benchmark and experiment usage
  std::vector<blst_scalar> encoded_scalars(num_pairs);
  encode_scalars(encoded_scalars, scalars_in, num_pairs, scalar_bits, c);

  // Use input variable scalar_bits rather than finding largest bit length
  size_t num_groups = (scalar_bits + c - 1) / c;

  // Variables for scalar bit window extraction
  limb_t bit_mask = (1 << c) - 1;
  bool   crosses_limbs = (((sizeof(limb_t) * 8) % c) != 0);
  size_t num_limbs = ceil((double)scalar_bits/8/sizeof(limb_t));

  size_t num_buckets = (1 << (c - 1));
  size_t bucket_mask = (1 << (c - 1)) - 1;

  // Has the result been set yet
  bool result_valid = false;
  *result = { { 0 }, { 0 }, { 0 } };
  blst_p1_ext inf = { { 0 }, { 0 }, { 0 }, { 0 } }; // Infinity

  std::vector<blst_p1_ext> buckets(num_buckets);

  // Loop through all windows
  for (size_t k = num_groups - 1; k <= num_groups; k--) {
    // Need to double result c times once set
    if (result_valid == true) {
      for (size_t i = 0; i < c; ++i) {
        blst_p1_double(result, result);
      }
    }

    // Set all buckets to infinity
    std::fill(buckets.begin(), buckets.end(), inf);

    // Current masks for bit window extraction
    size_t index = (k * c) / (sizeof(limb_t) * 8);
    size_t shift = (k * c) - (index * (sizeof(limb_t) * 8));
    limb_t mask  = bit_mask << shift;
    bool multi_limb = crosses_limbs &&
                      (shift > ((sizeof(limb_t) * 8) - c)) &&
                      (index < (num_limbs - 1));
    size_t shift_high = 0;
    limb_t mask_high  = 0;
    if (multi_limb) {
      shift_high = c - (shift - ((sizeof(limb_t) * 8) - c));
      mask_high  = (1 << (shift - ((sizeof(limb_t) * 8) - c))) - 1;
    }

    // Loop through all points and add associated point to corresponding bucket
    for (size_t i = 0; i < num_pairs; ++i) {
      size_t bucket = 0;

      // Determine bucket based on value of scalar bits in current window
      bucket = (encoded_scalars[i][index] & mask) >> shift;
      if (multi_limb) {
        bucket += (encoded_scalars[i][index + 1] & mask_high) << shift_high;
      }

      // If no bits are set then skip to next pair
      if (bucket == 0) {
        continue;
      }

      // Add or assign base to bucket value
      if ((bucket & num_buckets) == 0) {
        blst_p1_ext_add_or_double_affine(&(buckets[bucket - 1]),
                                         &(buckets[bucket - 1]),
                                         &(bases_in[i].p));
      }
      else {
        blst_p1_affine cur_base_neg;
        std::memcpy(&cur_base_neg, &(bases_in[i].p), sizeof(blst_p1_affine));
        blst_fp_cneg(cur_base_neg.Y, cur_base_neg.Y, true);
        blst_p1_ext_add_or_double_affine(&(buckets[bucket & bucket_mask]),
                                         &(buckets[bucket & bucket_mask]),
                                         &(cur_base_neg));
      }
    }

    blst_p1 cur_sum = { { 0 }, { 0 }, { 0 } };

    // Add all the buckets to the result
    blst_p1 cur_bucket;
    for (int i = num_buckets - 1; i >= 0; i--) {
      if (blst_p1_ext_is_inf(&(buckets[i])) == false) {
        blst_p1_from_extended_no_check(&cur_bucket, &(buckets[i]));
        blst_p1_add_or_double(&cur_sum, &cur_sum, &cur_bucket);
        result_valid = true;
      }
      if (result_valid == true) {
        blst_p1_add_or_double(result, result, &cur_sum);
      }
    }
  }
}
