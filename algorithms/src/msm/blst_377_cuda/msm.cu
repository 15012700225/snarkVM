#include "hip/hip_runtime.h"
#include "blst_377_ops.h"
#include <stdio.h>

template <size_t scalar_bits, size_t bit_width, size_t num_buckets>
__device__ void msm6_window(blst_p1* output, const blst_p1_affine* bases_in, const blst_scalar* scalars, size_t scalar_len) {
    blst_p1 buckets[num_buckets];
    for (size_t i = 0; i < num_buckets; ++i) {
        memcpy(&buckets[i], &BLS12_377_ZERO_PROJECTIVE, sizeof(blst_p1));
    }

    // Current masks for bit window extraction
    limb_t bit_mask = (1 << bit_width) - 1;
    bool crosses_limbs = (((sizeof(limb_t) * 8) % bit_width) != 0);
    size_t num_limbs = ceil((double)scalar_bits/8/sizeof(limb_t));

    limb_t index = (threadIdx.x * bit_width) / 64;
    size_t shift = (threadIdx.x * bit_width) - (index * (sizeof(limb_t) * 8));
    limb_t mask  = bit_mask << shift;
    bool multi_limb = crosses_limbs &&
                      (shift > ((sizeof(limb_t) * 8) - bit_width)) &&
                      (index < (num_limbs - 1));
    size_t shift_high = 0;
    limb_t mask_high  = 0;
    if (multi_limb) {
      shift_high = bit_width - (shift - ((sizeof(limb_t) * 8) - bit_width));
      mask_high  = (1 << (shift - ((sizeof(limb_t) * 8) - bit_width))) - 1;
    }

    // Loop through all points and add associated point to corresponding bucket
    for (size_t i = 0; i < scalar_len; ++i) {
        limb_t bucket = (scalars[i][index] & mask) >> shift;
        if (multi_limb) {
            bucket += (scalars[i][index + 1] & mask_high) << shift_high;
        }
        // printf("t%i: scalar %llu: bucket=%llu sindex=%llu scalar=%llu base_x=%llu base_y=%llu\n", threadIdx.x, i, bucket, index, scalars[i][index], bases_in[i].X[0], bases_in[i].Y[0]);

        // If no bits are set then skip to next pair
        if (bucket == 0) {
            continue;
        }

        // todo: cost of subtraction probably isnt worth it
        // limb_t prebucket = buckets[bucket - 1].X[0];
        blstv2_add_affine_to_projective(&(buckets[bucket - 1]), &(buckets[bucket - 1]), &(bases_in[i]));
        // printf("c-t%i: sindex=%llu ssindex=%llu scalar=%llu base_x=%llu output=%llu prebucket=%llu\n", threadIdx.x, i, index, scalars[i][index], bases_in[i].X[0], buckets[bucket - 1].X[0], prebucket);
    }

    blst_p1 running_sum;
    memcpy(&running_sum, &BLS12_377_ZERO_PROJECTIVE, sizeof(blst_p1));
    blst_p1 out;
    memcpy(&out, &BLS12_377_ZERO_PROJECTIVE, sizeof(blst_p1));

    for (int i = num_buckets - 1; i >= 0; i--) {
        // printf("c-t%i:pre: bucket %i: bx=%llu by=%llu bz=%llu\n", threadIdx.x, i, buckets[i].X[0], buckets[i].Y[0], buckets[i].Z[0]);
        blst_p1_affine affine;
        blst_p1_projective_into_affine(&affine, &(buckets[i]));
        // printf("c-t%i:affine: bucket %i: ax=%llu ay=%llu\n", threadIdx.x, i, affine.X[0], affine.Y[0]);
        blstv2_add_affine_to_projective(&running_sum, &running_sum, &affine);
        // printf("c-t%i:added: bucket %i: sx=%llu sy=%llu sz=%llu\n", threadIdx.x, i, running_sum.X[0], running_sum.Y[0], running_sum.Z[0]);
        blst_p1_add_or_double(&out, &out, &running_sum);
        // printf("c-t%i:committed: bucket %i: ox=%llu oy=%llu oz=%llu\n", threadIdx.x, i, out.X[0], out.Y[0], out.Z[0]);
    }
    memcpy(&output[threadIdx.x], &out, sizeof(blst_p1));
}

extern "C" __global__ void msm6_window_253_1(blst_p1* output, const blst_p1_affine* bases_in, const blst_scalar* scalars, size_t scalar_len) {
  msm6_window<253, 1, 1>(output, bases_in, scalars, scalar_len);
}
