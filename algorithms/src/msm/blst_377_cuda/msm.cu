#include "hip/hip_runtime.h"
#include "blst_377_ops.h"
#include <stdio.h>
#include <stdint.h>

static const uint32_t WINDOW_SIZE = 32;
static const uint32_t BLST_WIDTH = 253;

extern "C" __global__ void msm6_pixel(blst_p1* bucket_lists, const blst_p1_affine* bases_in, const blst_scalar* scalars, const uint32_t* window_lengths, const uint32_t window_count) {
    limb_t index = threadIdx.x / 64;
    size_t shift = threadIdx.x - (index * 64);
    // printf("c-%i:%i: running\n", threadIdx.x, blockIdx.x);

    blst_p1 bucket;
    memcpy(&bucket, &BLS12_377_ZERO_PROJECTIVE, sizeof(blst_p1));

    uint32_t window_start = WINDOW_SIZE * blockIdx.x;
    uint32_t window_end = window_start + window_lengths[blockIdx.x];
    // printf("c-%i:%i: windows %u -> %u\n", threadIdx.x, blockIdx.x, window_start, window_start + window_lengths[blockIdx.x]);
    for (uint32_t i = window_start; i < window_end; ++i) {
        // printf("c-%i:%i: doing %lu\n", threadIdx.x, blockIdx.x, i);
        limb_t bit = (scalars[i][index] >> shift) & 1;
        if (bit == 0) {
            continue;
        }
        blstv2_add_affine_to_projective(&bucket, &bucket, &(bases_in[i]));
    }

    memcpy(&bucket_lists[threadIdx.x * window_count + blockIdx.x], &bucket, sizeof(blst_p1));
}

extern "C" __global__ void msm6_collapse_rows(blst_p1* target, const blst_p1* bucket_lists, const uint32_t window_count) {
    blst_p1 temp_target;
    uint32_t base = threadIdx.x * window_count;
    uint32_t term = base + window_count;
    memcpy(&temp_target, &bucket_lists[base], sizeof(blst_p1));

    for (uint32_t i = base + 1; i < term; ++i) {
        blst_p1_add_or_double(&temp_target, &temp_target, &bucket_lists[i]);
    }
    
    memcpy(&target[threadIdx.x], &temp_target, sizeof(blst_p1));
}

// extern "C" __global__ void msm6_window_253_1(blst_p1* output, const blst_p1* bucket_list) {
//     blst_p1 running_sum;
//     memcpy(&running_sum, &BLS12_377_ZERO_PROJECTIVE, sizeof(blst_p1));
//     blst_p1 out;
//     memcpy(&out, &BLS12_377_ZERO_PROJECTIVE, sizeof(blst_p1));

//     // printf("c-t%i:pre: bucket %i: bx=%llu by=%llu bz=%llu\n", threadIdx.x, i, buckets[i].X[0], buckets[i].Y[0], buckets[i].Z[0]);
//     blst_p1_affine affine;
//     blst_p1_projective_into_affine(&affine, &bucket);
//     // printf("c-t%i:affine: bucket %i: ax=%llu ay=%llu\n", threadIdx.x, i, affine.X[0], affine.Y[0]);
//     blstv2_add_affine_to_projective(&running_sum, &running_sum, &affine);
//     // printf("c-t%i:added: bucket %i: sx=%llu sy=%llu sz=%llu\n", threadIdx.x, i, running_sum.X[0], running_sum.Y[0], running_sum.Z[0]);
//     blst_p1_add_or_double(&out, &out, &running_sum);
//     // printf("c-t%i:committed: bucket %i: ox=%llu oy=%llu oz=%llu\n", threadIdx.x, i, out.X[0], out.Y[0], out.Z[0]);
//     memcpy(&output[threadIdx.x], &out, sizeof(blst_p1));
// }

/*

extern "C" __global__ void msm6_window_253_1(blst_p1* output, const blst_p1_affine* bases_in, const blst_scalar* scalars, size_t scalar_len) {
    blst_p1 bucket;
    memcpy(&bucket, &BLS12_377_ZERO_PROJECTIVE, sizeof(blst_p1));

    limb_t index = threadIdx.x / 64;
    size_t shift = threadIdx.x - (index * 64);

    // Loop through all points and add associated point to corresponding bucket
    for (size_t i = 0; i < scalar_len; ++i) {
        limb_t bit = (scalars[i][index] >> shift) & 1;
        // printf("t%i: scalar %llu: bucket=%llu sindex=%llu scalar=%llu base_x=%llu base_y=%llu\n", threadIdx.x, i, bucket, index, scalars[i][index], bases_in[i].X[0], bases_in[i].Y[0]);

        // If no bits are set then skip to next pair
        if (bit == 0) {
            continue;
        }

        // todo: cost of subtraction probably isnt worth it
        // limb_t prebucket = buckets[bucket - 1].X[0];
        blstv2_add_affine_to_projective(&bucket, &bucket, &(bases_in[i]));
        // printf("c-t%i: sindex=%llu ssindex=%llu scalar=%llu base_x=%llu output=%llu prebucket=%llu\n", threadIdx.x, i, index, scalars[i][index], bases_in[i].X[0], buckets[bucket - 1].X[0], prebucket);
    }

    blst_p1 running_sum;
    memcpy(&running_sum, &BLS12_377_ZERO_PROJECTIVE, sizeof(blst_p1));
    blst_p1 out;
    memcpy(&out, &BLS12_377_ZERO_PROJECTIVE, sizeof(blst_p1));

    // printf("c-t%i:pre: bucket %i: bx=%llu by=%llu bz=%llu\n", threadIdx.x, i, buckets[i].X[0], buckets[i].Y[0], buckets[i].Z[0]);
    blst_p1_affine affine;
    blst_p1_projective_into_affine(&affine, &bucket);
    // printf("c-t%i:affine: bucket %i: ax=%llu ay=%llu\n", threadIdx.x, i, affine.X[0], affine.Y[0]);
    blstv2_add_affine_to_projective(&running_sum, &running_sum, &affine);
    // printf("c-t%i:added: bucket %i: sx=%llu sy=%llu sz=%llu\n", threadIdx.x, i, running_sum.X[0], running_sum.Y[0], running_sum.Z[0]);
    blst_p1_add_or_double(&out, &out, &running_sum);
    // printf("c-t%i:committed: bucket %i: ox=%llu oy=%llu oz=%llu\n", threadIdx.x, i, out.X[0], out.Y[0], out.Z[0]);
    memcpy(&output[threadIdx.x], &out, sizeof(blst_p1));
}

*/