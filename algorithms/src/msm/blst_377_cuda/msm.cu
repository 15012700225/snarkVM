#include "hip/hip_runtime.h"
#include "blst_377_ops.h"
#include <stdio.h>
#include <stdint.h>

static const uint32_t WINDOW_SIZE = 128;
static const uint32_t BLST_WIDTH = 253;

extern "C" __global__ void msm6_pixel(blst_p1* bucket_lists, const blst_p1_affine* bases_in, const blst_scalar* scalars, const uint32_t* window_lengths, const uint32_t window_count) {
    limb_t index = threadIdx.x / 64;
    size_t shift = threadIdx.x - (index * 64);
    limb_t mask = (limb_t) 1 << (limb_t) shift;

    blst_p1 bucket;
    memcpy(&bucket, &BLS12_377_ZERO_PROJECTIVE, sizeof(blst_p1));

    uint32_t window_start = WINDOW_SIZE * blockIdx.x;
    uint32_t window_end = window_start + window_lengths[blockIdx.x];

    uint32_t activated_bases[WINDOW_SIZE];
    uint32_t activated_base_index = 0;

    // printf("c-%i:%i: windows %u -> %u\n", threadIdx.x, blockIdx.x, window_start, window_start + window_lengths[blockIdx.x]);
    // we delay the actual additions to a second loop because it reduces warp divergence (20% practical gain)
    for (uint32_t i = window_start; i < window_end; ++i) {
        // printf("c-%i:%i: doing %lu\n", threadIdx.x, blockIdx.x, i);
        // limb_t bitw = (scalars[i][index] >> shift) & 1;
        limb_t bit = (scalars[i][index] & mask);
        if (bit == 0) {
            continue;
        }
        activated_bases[activated_base_index++] = i;
    }
    for (uint32_t i = 0; i < activated_base_index; ++i) {
        blst_add_affine_to_projective(&bucket, &bucket, &(bases_in[activated_bases[i]]));
    }

    memcpy(&bucket_lists[threadIdx.x * window_count + blockIdx.x], &bucket, sizeof(blst_p1));
}

extern "C" __global__ void msm6_collapse_rows(blst_p1* target, const blst_p1* bucket_lists, const uint32_t window_count) {
    blst_p1 temp_target;
    uint32_t base = threadIdx.x * window_count;
    uint32_t term = base + window_count;
    memcpy(&temp_target, &bucket_lists[base], sizeof(blst_p1));

    for (uint32_t i = base + 1; i < term; ++i) {
        blst_add_projective_to_projective(&temp_target, &temp_target, &bucket_lists[i]);
    }
    
    memcpy(&target[threadIdx.x], &temp_target, sizeof(blst_p1));
}
